﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cmath>
#include <ctime>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

struct Color_BGR
{
    int B, G, R;
};

struct Color_Lab
{
    float L, a, b;
};

Color_Lab BGR2Lab(Color_BGR x)
{
#define gamma(x) (((x) > 0.04045) ? std::pow(((x)+0.055f) / 1.055f, 2.4f) : ((x) / 12.92));

    const float param_13 = 1.0f / 3.0f;
    const float param_16116 = 16.0f / 116.0f;
    const float Xn = 0.950456f;
    const float Yn = 1.0f;
    const float Zn = 1.088754f;


    float RR = gamma(x.R / 255.0);
    float GG = gamma(x.G / 255.0);
    float BB = gamma(x.B / 255.0);

    float X, Y, Z, fX, fY, fZ;

    X = 0.4124564f * RR + 0.3575761f * GG + 0.1804375f * BB;
    Y = 0.2126729f * RR + 0.7151522f * GG + 0.0721750f * BB;
    Z = 0.0193339f * RR + 0.1191920f * GG + 0.9503041f * BB;

    X /= (Xn);
    Y /= (Yn);
    Z /= (Zn);

    if (Y > 0.008856f)
        fY = std::pow(Y, param_13);
    else
        fY = 7.787f * Y + param_16116;

    if (X > 0.008856f)
        fX = std::pow(X, param_13);
    else
        fX = 7.787f * X + param_16116;

    if (Z > 0.008856)
        fZ = std::pow(Z, param_13);
    else
        fZ = 7.787f * Z + param_16116;

    float L, a, b;

    L = 116.0f * fY - 16.0f;
    L = L > 0.0f ? L : 0.0f;
    a = 500.0f * (fX - fY);
    b = 200.0f * (fY - fZ);

    return { L,a,b };
}

hipError_t FindColorCuda(Color_BGR *src, float *ret,Color_Lab target,unsigned int size);

__global__ void FindColorCudaKernel(Color_BGR *src, float* ret, Color_Lab target)
{
    int i = blockIdx.x * 256 + threadIdx.x;

#define gamma(x) (((x) > 0.04045) ? pow(((x)+0.055f) / 1.055f, 2.4f) : ((x) / 12.92));

    const float param_13 = 1.0f / 3.0f;
    const float param_16116 = 16.0f / 116.0f;
    const float Xn = 0.950456f;
    const float Yn = 1.0f;
    const float Zn = 1.088754f;

    float RR = gamma(src[i].R / 255.0);
    float GG = gamma(src[i].G / 255.0);
    float BB = gamma(src[i].B / 255.0);

    float X, Y, Z, fX, fY, fZ;

    X = 0.4124564f * RR + 0.3575761f * GG + 0.1804375f * BB;
    Y = 0.2126729f * RR + 0.7151522f * GG + 0.0721750f * BB;
    Z = 0.0193339f * RR + 0.1191920f * GG + 0.9503041f * BB;

    X /= (Xn);
    Y /= (Yn);
    Z /= (Zn);

    if (Y > 0.008856f)
        fY = pow(Y, param_13);
    else
        fY = 7.787f * Y + param_16116;

    if (X > 0.008856f)
        fX = pow(X, param_13);
    else
        fX = 7.787f * X + param_16116;

    if (Z > 0.008856)
        fZ = pow(Z, param_13);
    else
        fZ = 7.787f * Z + param_16116;

    float L, a, b;

    L = 116.0f * fY - 16.0f;
    L = L > 0.0f ? L : 0.0f;
    a = 500.0f * (fX - fY);
    b = 200.0f * (fY - fZ);

    ret[i] = sqrt((L - target.L) * (L - target.L) + (a - target.a) * (a - target.a) + (b - target.b) * (b - target.b));
}

Color_BGR src_mat[1024 * 1024];
float ret_mat[1024 * 1024];


int main()
{
    for (int i = 0; i < 1024 * 1024; i++)
    {
        src_mat[i] = { std::rand() % 256,std::rand() % 256, std::rand() % 256 };
    }
    //Pre Run for Best Speed
    hipError_t cudaStatus = FindColorCuda(src_mat, ret_mat, BGR2Lab({ 190,35,41 }), 1024 * 1024);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "FindColorCuda failed!");
        return 1;
    }

    int st = clock();
    // Add vectors in parallel.
    cudaStatus = FindColorCuda(src_mat, ret_mat, BGR2Lab({190,35,41}), 1024 * 1024);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "FindColorCuda failed!");
        return 1;
    }
    printf("Cost: %d\n", clock() - st);

    int count = 0;
    for (int i = 0; i < 1024*1024 ; i++)
    {
        if (ret_mat[i] < 2)
            count++;
    }
    printf("%d", count);
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

//Helper
hipError_t FindColorCuda(Color_BGR* src, float* ret, Color_Lab target, unsigned int size)
{
    Color_BGR* dev_src = nullptr;
    float* dev_ret = nullptr;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_src, size * sizeof(Color_BGR));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_ret, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_src, src, size * sizeof(Color_BGR), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    FindColorCudaKernel <<<size/256, 256 >>> (dev_src,dev_ret,target);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "FindColorCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy( ret, dev_ret, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_ret);
    hipFree(dev_src);

    return cudaStatus;
}